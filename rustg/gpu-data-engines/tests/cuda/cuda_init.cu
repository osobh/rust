/**
 * CUDA Runtime Initialization for Tests
 * Provides proper GPU context initialization for test harness
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

// Global variable to track CUDA initialization
static bool cuda_initialized = false;

extern "C" {
    /**
     * Initialize CUDA runtime
     * Returns 0 on success, error code otherwise
     */
    int cuda_init() {
        if (cuda_initialized) {
            return 0;  // Already initialized
        }
        
        // Set device to 0 (first GPU)
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to set CUDA device: %s\n", hipGetErrorString(err));
            return (int)err;
        }
        
        // Force runtime initialization
        hipFree(0);
        
        // Reset any prior errors
        hipGetLastError();
        
        cuda_initialized = true;
        return 0;
    }
    
    /**
     * Check if CUDA is initialized
     */
    bool cuda_is_initialized() {
        return cuda_initialized;
    }
    
    /**
     * Get number of available CUDA devices
     */
    int cuda_device_count() {
        int count = 0;
        hipError_t err = hipGetDeviceCount(&count);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to get CUDA device count: %s\n", hipGetErrorString(err));
            return 0;
        }
        return count;
    }
    
    /**
     * Reset CUDA device
     */
    int cuda_reset() {
        hipError_t err = hipDeviceReset();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to reset CUDA device: %s\n", hipGetErrorString(err));
            return (int)err;
        }
        return 0;
    }
}