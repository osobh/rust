#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>
#include <cstdint>
#include <climits>

// Global memory pool handle for async allocations (CUDA 13.0)
static hipMemPool_t global_mem_pool = nullptr;
static hipStream_t default_stream = nullptr;

extern "C" {

// Initialize CUDA runtime with memory pools (CUDA 13.0)
int cuda_initialize() {
  // Set device 0 as default
  hipError_t err = hipSetDevice(0);
  if (err != hipSuccess) {
    return static_cast<int>(err);
  }
  
  // Create default stream for async operations
  err = hipStreamCreate(&default_stream);
  if (err != hipSuccess) {
    return static_cast<int>(err);
  }
  
  // Get the default memory pool for the current device (CUDA 13.0)
  int device;
  hipGetDevice(&device);
  err = hipDeviceGetDefaultMemPool(&global_mem_pool, device);
  if (err != hipSuccess) {
    // Fallback: create a custom memory pool if default not available
    hipMemPoolProps pool_props = {};
    pool_props.allocType = hipMemAllocationTypePinned;
    pool_props.handleTypes = hipMemHandleTypeNone;
    pool_props.location.type = hipMemLocationTypeDevice;
    pool_props.location.id = device;
    
    err = hipMemPoolCreate(&global_mem_pool, &pool_props);
    if (err != hipSuccess) {
      return static_cast<int>(err);
    }
  }
  
  // Set memory pool attributes for better performance
  uint64_t threshold = UINT64_MAX;  // No release threshold
  hipMemPoolSetAttribute(global_mem_pool, hipMemPoolAttrReleaseThreshold, &threshold);
  
  return 0;
}

// Clean up CUDA runtime and memory pools
int cuda_cleanup() {
  hipError_t err = hipSuccess;
  
  // Destroy custom memory pool if created
  if (global_mem_pool != nullptr) {
    err = hipMemPoolDestroy(global_mem_pool);
    global_mem_pool = nullptr;
  }
  
  // Destroy stream
  if (default_stream != nullptr) {
    err = hipStreamDestroy(default_stream);
    default_stream = nullptr;
  }
  
  err = hipDeviceReset();
  return static_cast<int>(err);
}

// Get number of CUDA devices
int cuda_get_device_count() {
  int count = 0;
  hipError_t err = hipGetDeviceCount(&count);
  if (err != hipSuccess) {
    return -1;
  }
  return count;
}

// Get device memory size
size_t cuda_get_device_memory_size(int device) {
  hipDeviceProp_t prop;
  hipError_t err = hipGetDeviceProperties(&prop, device);
  if (err != hipSuccess) {
    return 0;
  }
  
  // Reserve some memory for system use (90% of total)
  return (size_t)(prop.totalGlobalMem * 0.9);
}

// Allocate device memory (legacy for compatibility)
void* cuda_malloc_device(size_t size) {
  void* ptr = nullptr;
  hipError_t err = hipMalloc(&ptr, size);
  if (err != hipSuccess) {
    return nullptr;
  }
  
  // Initialize memory to zero
  err = hipMemset(ptr, 0, size);
  if (err != hipSuccess) {
    hipFree(ptr);
    return nullptr;
  }
  
  return ptr;
}

// Allocate device memory asynchronously (CUDA 13.0 - preferred)
void* cuda_malloc_async(size_t size, hipStream_t stream) {
  void* ptr = nullptr;
  hipStream_t str = (stream != nullptr) ? stream : default_stream;
  
  // Use async allocation from memory pool
  hipError_t err = hipMallocAsync(&ptr, size, str);
  if (err != hipSuccess) {
    return nullptr;
  }
  
  // Initialize memory to zero asynchronously
  err = hipMemsetAsync(ptr, 0, size, str);
  if (err != hipSuccess) {
    hipFreeAsync(ptr, str);
    return nullptr;
  }
  
  return ptr;
}

// Allocate host-pinned memory with async API (CUDA 13.0)
void* cuda_malloc_host_async(size_t size) {
  void* ptr = nullptr;
  
  // Use CU_MEM_LOCATION_TYPE_HOST for host-pinned allocation
  hipError_t err = hipHostMalloc(&ptr, size, hipHostMallocDefault);
  if (err != hipSuccess) {
    return nullptr;
  }
  
  // Initialize to zero
  memset(ptr, 0, size);
  return ptr;
}

// Free device memory (legacy for compatibility)
int cuda_free_device(void* ptr) {
  if (ptr == nullptr) {
    return 0;
  }
  hipError_t err = hipFree(ptr);
  return static_cast<int>(err);
}

// Free device memory asynchronously (CUDA 13.0 - preferred)
int cuda_free_async(void* ptr, hipStream_t stream) {
  if (ptr == nullptr) {
    return 0;
  }
  hipStream_t str = (stream != nullptr) ? stream : default_stream;
  hipError_t err = hipFreeAsync(ptr, str);
  return static_cast<int>(err);
}

// Free host-pinned memory
int cuda_free_host(void* ptr) {
  if (ptr == nullptr) {
    return 0;
  }
  hipError_t err = hipHostFree(ptr);
  return static_cast<int>(err);
}

// Copy memory from host to device (synchronous)
int cuda_memcpy_host_to_device(void* dst, const void* src, size_t size) {
  hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
  return static_cast<int>(err);
}

// Copy memory from host to device asynchronously (CUDA 13.0 - preferred)
int cuda_memcpy_host_to_device_async(void* dst, const void* src, size_t size, hipStream_t stream) {
  hipStream_t str = (stream != nullptr) ? stream : default_stream;
  hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, str);
  return static_cast<int>(err);
}

// Copy memory from device to host (synchronous)
int cuda_memcpy_device_to_host(void* dst, const void* src, size_t size) {
  hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
  return static_cast<int>(err);
}

// Copy memory from device to host asynchronously (CUDA 13.0 - preferred)
int cuda_memcpy_device_to_host_async(void* dst, const void* src, size_t size, hipStream_t stream) {
  hipStream_t str = (stream != nullptr) ? stream : default_stream;
  hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, str);
  return static_cast<int>(err);
}

// Synchronize device
int cuda_synchronize() {
  hipError_t err = hipDeviceSynchronize();
  return static_cast<int>(err);
}

// Get last error
int cuda_get_last_error() {
  hipError_t err = hipGetLastError();
  return static_cast<int>(err);
}

// Get error string
const char* cuda_get_error_string(int error) {
  return hipGetErrorString(static_cast<hipError_t>(error));
}

} // extern "C"