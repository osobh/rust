#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>

extern "C" {

// Initialize CUDA runtime
int cuda_initialize() {
  // Set device 0 as default
  hipError_t err = hipSetDevice(0);
  if (err != hipSuccess) {
    return static_cast<int>(err);
  }
  
  // Reset device to clear any previous state
  err = hipDeviceReset();
  return static_cast<int>(err);
}

// Clean up CUDA runtime
int cuda_cleanup() {
  hipError_t err = hipDeviceReset();
  return static_cast<int>(err);
}

// Get number of CUDA devices
int cuda_get_device_count() {
  int count = 0;
  hipError_t err = hipGetDeviceCount(&count);
  if (err != hipSuccess) {
    return -1;
  }
  return count;
}

// Get device memory size
size_t cuda_get_device_memory_size(int device) {
  hipDeviceProp_t prop;
  hipError_t err = hipGetDeviceProperties(&prop, device);
  if (err != hipSuccess) {
    return 0;
  }
  
  // Reserve some memory for system use (90% of total)
  return (size_t)(prop.totalGlobalMem * 0.9);
}

// Allocate device memory
void* cuda_malloc_device(size_t size) {
  void* ptr = nullptr;
  hipError_t err = hipMalloc(&ptr, size);
  if (err != hipSuccess) {
    return nullptr;
  }
  
  // Initialize memory to zero
  err = hipMemset(ptr, 0, size);
  if (err != hipSuccess) {
    hipFree(ptr);
    return nullptr;
  }
  
  return ptr;
}

// Free device memory
int cuda_free_device(void* ptr) {
  if (ptr == nullptr) {
    return 0;
  }
  hipError_t err = hipFree(ptr);
  return static_cast<int>(err);
}

// Copy memory from host to device
int cuda_memcpy_host_to_device(void* dst, const void* src, size_t size) {
  hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
  return static_cast<int>(err);
}

// Copy memory from device to host
int cuda_memcpy_device_to_host(void* dst, const void* src, size_t size) {
  hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
  return static_cast<int>(err);
}

// Synchronize device
int cuda_synchronize() {
  hipError_t err = hipDeviceSynchronize();
  return static_cast<int>(err);
}

// Get last error
int cuda_get_last_error() {
  hipError_t err = hipGetLastError();
  return static_cast<int>(err);
}

// Get error string
const char* cuda_get_error_string(int error) {
  return hipGetErrorString(static_cast<hipError_t>(error));
}

} // extern "C"